#include <chrono>
#include <iostream>
#include <string>

#include "cnn.cuh"
#include "utils.cuh"
#include "cnn_seq.cuh"
#include "../cnn_gpu.cuh"

using std::chrono::duration_cast;
using std::chrono::microseconds;
using std::chrono::steady_clock;
using std::clog;
using std::string;

void cnn_gpu_wrapper(float* g_input,float* g_weight,float* g_bias,float* g_output) {
  // get params
  auto block = get_param("BLOCK");
  auto grid = get_param("GRID");
  std::clog << "Using thread block dims: " << block.x << ' ' << block.y << ' ' << block.z << '\n';
  std::clog << "Using grid dims: " << grid.x << ' ' << grid.y << ' ' << grid.z << '\n';
  
  // set device
  hipSetDevice(0);
  
  // execute kernel
  cnn_gpu<<<grid, block>>>(g_input, g_weight, g_bias, g_output);
  CUDA_CHECK(hipDeviceSynchronize()); // wait until kernel is completely finished
}

int main(int argc, char** argv) {

  // sizes are known ahead of time for this particular example
  auto input_size = kNum * kInImSize * kInImSize * sizeof(float);
  auto weight_size = kNum * kNum * kKernel * kKernel * sizeof(float);
  auto bias_size = kNum * sizeof(float);
  auto output_size = kNum * kOutImSize * kOutImSize * sizeof(float);

  // allocate memory on heap to avoid stack overflow
  float* input = static_cast<float*>(malloc(input_size));
  float* weight = static_cast<float*>(malloc(weight_size));
  float* bias = static_cast<float*>(malloc(bias_size));
  float* output = static_cast<float*>(malloc(output_size));

  if (argc > 2) {
    clog << "Usage: " << argv[0] << " [data dir]\n";
    return EXIT_FAILURE;
  }

  // load data
  const string data_dir = argc == 2 ? string(argv[1]) + "/" : "lib/data/";
  LoadData(data_dir, input, weight, bias);

  // create device memory
  float* g_input, *g_weight, *g_bias, *g_output;
  if (!getenv("SEQUENTIAL")) {
    clog << "Create device memory\n";
    hipMalloc((float**)&g_input, input_size);
    hipMalloc((float**)&g_weight, weight_size);
    hipMalloc((float**)&g_bias, bias_size);
    hipMalloc((float**)&g_output, output_size);
  }

  // transfer to global memory
  if (!getenv("SEQUENTIAL")) {
    clog << "Transfer to global memory\n";
    hipMemcpy(g_input, input, input_size, hipMemcpyHostToDevice);
    hipMemcpy(g_weight, weight, weight_size, hipMemcpyHostToDevice);
    hipMemcpy(g_bias, bias, bias_size, hipMemcpyHostToDevice);
  }

  // invoke CNN kernel
  clog << "Invoke CNN computation kernel\n";
  auto begin = steady_clock::now();
  if (getenv("SEQUENTIAL")) {
    cnn_seq(input, weight, bias, output);
  } else {
    cnn_gpu_wrapper(g_input, g_weight, g_bias, g_output);
  }
  auto end = steady_clock::now();
  uint64_t run_time_us = duration_cast<microseconds>(end - begin).count();
  float gflops = float(kNum) * kNum * kImSize * kImSize * kKernel * kKernel * 2
    / (run_time_us * 1e3);
  clog << "Time: " << run_time_us * 1e-6 << " s\n";
  clog << "Perf: " << gflops << " GFlops\n";

  // get the data back
  if (!getenv("SEQUENTIAL")) {
    hipMemcpy(output, g_output, output_size, hipMemcpyDeviceToHost);
  }

  // verify correctness
  int error = Verify(data_dir, output, output_size);
  if (error != 0) {
    clog << "Found " << error << " error" << (error > 1 ? "s\n" : "\n");
    clog << "FAIL\n";
    return EXIT_FAILURE;
  } else {
    clog << "PASS\n";
    return EXIT_SUCCESS;
  }
}
