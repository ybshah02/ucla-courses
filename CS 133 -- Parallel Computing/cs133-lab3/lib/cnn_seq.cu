#include "hip/hip_runtime.h"
#include "cnn.cuh"
#include "cnn_seq.cuh"

// Sequential CNN implementation
void cnn_seq(
    const float *input,
    const float *weight,
    const float *bias,
    float *output
  ) {

  // Allocate memory on heap to avoid stack overflow.
  auto c_size = kNum * kImSize * kImSize * sizeof(float);
  float *C = static_cast<float*>(malloc(c_size));

  // Bias
  for (int i = 0; i < kNum; ++i) {
    for (int h = 0; h < kImSize; ++h) {
      for (int w = 0; w < kImSize; ++w) {
        C(i,h,w) = bias[i];
      }
    }
  }

  // Convolution
  for (int i = 0; i < kNum; ++i) {
    for (int j = 0; j < kNum; ++j) {
      for (int h = 0; h < kImSize; ++h) {
        for (int w = 0; w < kImSize; ++w) {
          for (int p = 0; p < kKernel; ++p) {
            for (int q = 0; q < kKernel; ++q)
              C(i,h,w) += weight(i,j,p,q) * input(j,h+p,w+q);
          }
        }
      }
    }
  }

  // ReLU
  for (int i = 0; i < kNum; ++i) {
    for (int h = 0; h < kImSize; ++h) {
      for (int w = 0; w < kImSize; ++w) {
        C(i,h,w) = max(0.f, C(i,h,w));
      }
    }
  }

  // Max pooling
  for (int i = 0; i < kNum; ++i) {
    for (int h = 0; h < kOutImSize; ++h) {
      for (int w = 0; w < kOutImSize; ++w) {
        output(i,h,w) = max(
            max(C(i, h*2, w*2  ), C(i, h*2+1, w*2  )),
            max(C(i, h*2, w*2+1), C(i, h*2+1, w*2+1)));
      }
    }
  }

  delete C;
}