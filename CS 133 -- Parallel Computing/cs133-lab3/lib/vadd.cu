#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <random>

#include "utils.cuh"

using std::clog;
using std::endl;

// VADD kernel
__global__ void vadd_gpu(const float* g_a, const float* g_b, float* g_c, uint64_t n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    g_c[i] = g_a[i] + g_b[i];
  }
}

int main(int argc, const char* argv[]) {
  const uint64_t n = 1 << 20;
  static float a[n];
  static float b[n];
  static float c[n];
  static float c_base[n];

  // generate data
  std::default_random_engine generator(
      std::chrono::steady_clock::now().time_since_epoch().count());
  std::uniform_real_distribution<float> distribution(0.f, 1.f);
  for (uint64_t i = 0; i < n; ++i) {
    a[i] = distribution(generator);
    b[i] = distribution(generator);
    c_base[i] = a[i] + b[i];
  }

  // create device memory
  float* g_a, *g_b, *g_c;
  clog << "Create device memory\n";
  hipMalloc((float**)&g_a, n*sizeof(float));
  hipMalloc((float**)&g_b, n*sizeof(float));
  hipMalloc((float**)&g_c, n*sizeof(float));

  // transfer to global memory
  clog << "Transfer to global memory\n";
  hipMemcpy(g_a, a, n*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(g_b, b, n*sizeof(float), hipMemcpyHostToDevice);

  // set params
  auto block = get_param("BLOCK");
  auto grid = get_param("GRID");
  std::clog << "Using thread block dims: " << block.x << ' ' << block.y << ' ' << block.z << '\n';
  std::clog << "Using grid dims: " << grid.x << ' ' << grid.y << ' ' << grid.z << '\n';

  // set device
  hipSetDevice(0);
  
  // execute kernel
  clog << "Invoke VADD computation kernel\n";
  vadd_gpu<<<grid, block>>>(g_a, g_b, g_c, n);
  CUDA_CHECK(hipDeviceSynchronize()); // wait until kernel is completely finished

  // get back the data
  hipMemcpy(c, g_c, n*sizeof(float), hipMemcpyDeviceToHost);

  // check correctness
  for (uint64_t i = 0; i < n; ++i) {
    if (c[i] != c_base[i]) {
      clog << "FAIL" << i << endl;
      return 1;
    }
  }
  clog << "PASS" << endl;
  return 0;
}