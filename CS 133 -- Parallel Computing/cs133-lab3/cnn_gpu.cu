#include "hip/hip_runtime.h"
// Header inclusions, if any...
#include "lib/cnn.cuh"
#include "cnn_gpu.cuh"

__global__ void cnn_gpu(
    float *input,
    float *weight,
    float *bias,
    float *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.z * blockDim.z + threadIdx.z;

    h *= 2;
    w *= 2;

    if (i < kNum && h < kImSize && w < kImSize) {
        float conv[4] = {bias[i], bias[i], bias[i], bias[i]};

        for (int j = 0; j < kNum; ++j) {
            for (int p = 0; p < kKernel; ++p) {
                for (int q = 0; q < kKernel; ++q) {
                    if (h + p < kInImSize && w + q < kInImSize) {
                        conv[0] += weight(i, j, p, q) * input(j, h+p, w+q);
                        conv[1] += weight(i, j, p, q) * input(j, h+1+p, w+q);
                        conv[2] += weight(i, j, p, q) * input(j, h+p, w+1+q);
                        conv[3] += weight(i, j, p, q) * input(j, h+1+p, w+1+q);
                    }
                }
            }
        }

        float final_conv = max(max(conv[0], conv[1]), max(conv[2], conv[3]));
        if (h / 2 < kOutImSize && w / 2 < kOutImSize) {
            output(i, h / 2, w / 2) = max(0.f, final_conv);
        }
    }
}
